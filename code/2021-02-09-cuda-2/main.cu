#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void square(unsigned* matrix, unsigned* result, unsigned matrixsize)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned jj = 0; jj < matrixsize; ++jj) {
        for (unsigned kk = 0; kk < matrixsize; ++kk) {
            result[id * matrixsize + jj] +=
                matrix[id * matrixsize + kk] * matrix[kk * matrixsize + jj];
        }
    }
}

#define N 1000000
#define M 1000

int main(void)
{
    unsigned* hmatrix = (unsigned*)malloc(N * sizeof(unsigned));
    for (unsigned i = 0; i < N; ++i) {
        hmatrix[i] = i % 10;
    }

    unsigned* hresult = (unsigned*)malloc(N * sizeof(unsigned));

    unsigned* dmatrix;
    unsigned* dresult;

    hipMalloc(&dmatrix, N * sizeof(unsigned));
    hipMalloc(&dresult, N * sizeof(unsigned));

    hipMemcpy(dmatrix, hmatrix, N * sizeof(unsigned), hipMemcpyHostToDevice);

    dim3 grid(M, 1, 1);
    dim3 block(M, 1, 1);

    square<<<grid, block>>>(dmatrix, dresult, M);
    hipMemcpy(hresult, dresult, N * sizeof(unsigned), hipMemcpyDeviceToHost);

    free(hmatrix);
    free(hresult);

    hipFree(dmatrix);
    hipFree(dresult);

    return 0;
}
